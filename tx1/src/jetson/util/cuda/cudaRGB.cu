#include "hip/hip_runtime.h"
/*
 * http://github.com/dusty-nv
 */

#include "cudaRGB.h"

//-------------------------------------------------------------------------------------------------------------------------

__global__ void RGBToRGBAf(uchar3* srcImage,
                           float4* dstImage,
                           uint32_t width,       uint32_t height)
{
    int x, y, pixel;

    x = (blockIdx.x * blockDim.x) + threadIdx.x;
    y = (blockIdx.y * blockDim.y) + threadIdx.y;
	
    pixel = y * width + x;

    if (x >= width)
        return; 

    if (y >= height)
        return;

//	printf("cuda thread %i %i  %i %i pixel %i \n", x, y, width, height, pixel);
		
	const float  s  = 1.0f;
	const uchar3 px = srcImage[pixel];
	
	dstImage[pixel] = make_float4(px.x * s, px.y * s, px.z * s, 255.0f * s);
}

hipError_t cudaRGBToRGBAf( uchar3* srcDev, float4* destDev, size_t width, size_t height )
{
	if( !srcDev || !destDev )
		return hipErrorInvalidDevicePointer;

	const dim3 blockDim(8,8,1);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y), 1);

	RGBToRGBAf<<<gridDim, blockDim>>>( srcDev, destDev, width, height );
	
	return CUDA(hipGetLastError());
}

